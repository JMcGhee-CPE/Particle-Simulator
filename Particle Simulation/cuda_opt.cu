#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <vector>

#include <chrono>
#include <ctime>
#include <stdio.h>

#ifdef __APPLE_CC__
#include <GL/glut.h>
#else
#include <GL/glut.h>
#endif
#include "atoms.cu"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define THREADS_PER_BLOCK 512

__global__
void initalize_atoms(Atom** d_atoms)
{

  int linearIdx = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
  
  hiprandState_t state;

  /* we have to initialize the state */
  hiprand_init(0, /* the seed controls the sequence of random values that are produced */
              0, /* the sequence number is only important with multiple cores */
              linearIdx * 5, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              &state);


  if(linearIdx < NUMBER_OF_ATOMS){
    Atom* new_atom = (Atom*)malloc(sizeof(Atom));

    int x_position_i = ((2*linearIdx*ATOM_RADIUS*ATOM_SPACING) + ATOM_RADIUS) * 10000;

    int y_position_i = floor(((double)x_position_i) / (2*STARTING_WIDTH * 100)) * 2 * ATOM_RADIUS*ATOM_SPACING * 10000;

    x_position_i %= 200*STARTING_WIDTH;

    double x_position = (x_position_i - (STARTING_WIDTH * 100)) / 10000.0;
    double y_position = (y_position_i - (STARTING_HEIGHT * 100)) / 10000.0;

    glm::vec2 startingPosition;
    startingPosition = glm::vec2( x_position, y_position);

    glm::vec2 startingVelocity;
    startingVelocity = glm::vec2(((hiprand(&state) % 20000) / 1000.00) - 10,((hiprand(&state) % 20000) / 1000.00) - 10);

    new_atom->coords = startingPosition;
    new_atom->vel = startingVelocity;

    new_atom->color[0] = (hiprand(&state) % 100) / (float)100.0;
    new_atom->color[1] = (hiprand(&state) % 100) / (float)100.0;
    new_atom->color[2] = (hiprand(&state) % 100) / (float)100.0;

    d_atoms[linearIdx] = new_atom;

    //printf("Idx = %d; Coords: { %f, %f }; Vel: { %f, %f }\n", linearIdx, d_atoms[linearIdx]->coords[0], d_atoms[linearIdx]->coords[1], d_atoms[linearIdx]->vel[0], d_atoms[linearIdx]->vel[1]);
   
  }
}

__global__
void move_atoms(Atom** d_atoms)
{
  int linearIdx = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
  if(linearIdx < NUMBER_OF_ATOMS){
   float time = 1.0 / 60;
    if (d_atoms[linearIdx]->coords[1] >= (STARTING_HEIGHT / 100) || d_atoms[linearIdx]->coords[1] <= -(STARTING_HEIGHT / 100)){
      d_atoms[linearIdx]->vel *= glm::vec2(1,-1);
      if(d_atoms[linearIdx]->coords[1] >= (STARTING_HEIGHT / 100)) {
        d_atoms[linearIdx]->coords[1] = (STARTING_HEIGHT / 100);
      } else {
        d_atoms[linearIdx]->coords[1] = -(STARTING_HEIGHT / 100);
      }
    }
    if (d_atoms[linearIdx]->coords[0] >= (STARTING_WIDTH / 100) || d_atoms[linearIdx]->coords[0] <= -(STARTING_WIDTH / 100)){
      d_atoms[linearIdx]->vel *= glm::vec2(-1,1);
      if(d_atoms[linearIdx]->coords[0] >= (STARTING_WIDTH / 100)) {
        d_atoms[linearIdx]->coords[0] = (STARTING_WIDTH / 100);
      } else {
        d_atoms[linearIdx]->coords[0] = -(STARTING_WIDTH / 100);
      }
    }
    glm::vec2 displacement = d_atoms[linearIdx]->vel;
    displacement *= time;

    d_atoms[linearIdx]->coords += displacement;
  }
}

__global__
void collide_atoms(Atom** d_atoms)
{
  int linearIdx = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
  if(linearIdx < NUMBER_OF_ATOMS){

    glm::vec2 new_vector(0,0);
    d_atoms[linearIdx]->new_pos = d_atoms[linearIdx]->coords;
    int num_collisions = 0;
    for(int i = 0; i < NUMBER_OF_ATOMS; i++){
      bool is_current_atom = d_atoms[i] != d_atoms[linearIdx];
      //printf("Current position atom 1 = { %f, %f}; atom 2 = { %f, %f}\n", d_atoms[i]->coords[0], d_atoms[i]->coords[1], d_atoms[linearIdx]->coords[0], d_atoms[linearIdx]->coords[1]);
      float dist = sqrtf( powf(d_atoms[i]->coords[0] - d_atoms[linearIdx]->coords[0], 2) + powf(d_atoms[i]->coords[1] - d_atoms[linearIdx]->coords[1], 2));
      if(is_current_atom && (dist <= (2 * ATOM_RADIUS))){

        // Formula for elastic collisions https://williamecraver.wixsite.com/elastic-equations

        glm::vec2 position_vector = d_atoms[linearIdx]->coords;
        position_vector -= d_atoms[i]->coords;

        glm::vec2 unit_vector = position_vector;
        unit_vector /= glm::length(position_vector);


        glm::vec2 offset_vector = unit_vector;
        offset_vector *= (2 * ATOM_RADIUS * COLLISION_MOVE);

        glm::vec2 velocity_difference = d_atoms[linearIdx]->vel;
        velocity_difference -= d_atoms[i]->vel;

        float dotproduct = glm::dot(velocity_difference, position_vector);

        glm::vec2 contribution = position_vector;
        contribution *= -dotproduct / powf( glm::length(position_vector), 2);

        contribution += d_atoms[linearIdx]->vel;

        new_vector += contribution;

        d_atoms[linearIdx]->new_pos += offset_vector;

        num_collisions++;
      }
    }

    if(num_collisions > 0){
      //std::cout << "Num Collisions = " << num_collisions << " \n";
      new_vector /= num_collisions;
      //std::cout << "New Vector = {" << new_vector[0] << ", " << new_vector[1] << "} \n";
      d_atoms[linearIdx]->new_vel = new_vector;
      //std::cout << "New vector - { " << new_vel[0] << ", " << new_vel[1] << " }\n";
      d_atoms[linearIdx]->update_vel = true;
    }

    // Else no collisions, velocity unchanged

  }
}

__global__
void update_atoms(Atom** d_atoms)
{
  int linearIdx = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
  if(linearIdx < NUMBER_OF_ATOMS){
    if(d_atoms[linearIdx]->update_vel){
      //std::cout << "velocity updated!\n";
      d_atoms[linearIdx]->vel = d_atoms[linearIdx]->new_vel;
      d_atoms[linearIdx]->coords = d_atoms[linearIdx]->new_pos;
      d_atoms[linearIdx]->update_vel = false;
    }
  }
}

__global__
void prepare_draw(Atom** d_atoms, Atom* d_atoms_ref)
{
  int linearIdx = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
  if(linearIdx < NUMBER_OF_ATOMS){
    memcpy(&d_atoms_ref[linearIdx], d_atoms[linearIdx],sizeof(Atom));
  }
}

void reshape(GLint w, GLint h) {

  glViewport(0, 0, w, h);   //This sets up the viewport so that the coordinates (0, 0) are at the top left of the window

  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();

  gluPerspective(65.0, GLfloat(w)/GLfloat(h), 1.0, 20.0);
  gluLookAt( 0,0,10, 0,0,0, 0,1,0);

  //Set up the orthographic projection so that coordinates (0, 0) are in the top left
  //and the minimum and maximum depth is -10 and 10. To enable depth just put in
  //glEnable(GL_DEPTH_TEST)

  glutPostRedisplay();

}

std::chrono::system_clock::time_point start;
    // Some computation here
std::chrono::system_clock::time_point end;

void timer(int time){

    end = std::chrono::system_clock::now();

    std::chrono::duration<double> elapsed_seconds = end-start;

    std::cout << "FPS = " <<  1/ elapsed_seconds.count() << "\n";

    glutPostRedisplay();

    glutTimerFunc( (1.0 / 60) * 1000, timer, time);

    start = std::chrono::system_clock::now();
}


void special(int key, int, int) {
  //return;
  switch (key) {
    //case GLUT_KEY_RIGHT:  glutPostRedisplay(); break;
    default: return;
  }
}

Atom **atoms, **d_atoms, *atom_refs, *d_atom_ref;

// Clears the current window and draws a triangle.
void display() {

  int number_of_blocks = (int)ceil(NUMBER_OF_ATOMS / (float)THREADS_PER_BLOCK);

  move_atoms<<< number_of_blocks, THREADS_PER_BLOCK>>>(d_atoms);

  hipDeviceSynchronize();

  collide_atoms<<< number_of_blocks, THREADS_PER_BLOCK>>>(d_atoms);

  hipDeviceSynchronize();

  update_atoms<<< number_of_blocks, THREADS_PER_BLOCK>>>(d_atoms);

  hipDeviceSynchronize();

  prepare_draw<<< number_of_blocks, THREADS_PER_BLOCK>>>(d_atoms, d_atom_ref);

  hipMemcpy(atom_refs, d_atom_ref, NUMBER_OF_ATOMS * sizeof(Atom), hipMemcpyDeviceToHost);

  glClear(GL_COLOR_BUFFER_BIT);

  glMatrixMode(GL_MODELVIEW);

  for(int i = 0; i < NUMBER_OF_ATOMS; i++){
    //atom_refs[i].print();
    atom_refs[i].draw();
  }   

  // Flush drawing command buffer to make drawing happen as soon as possible.
  glFlush();
}

int main(int argc, char** argv)
{

  atoms = (Atom**)malloc(NUMBER_OF_ATOMS* sizeof(Atom*));
  atom_refs = (Atom*)malloc(NUMBER_OF_ATOMS * sizeof(Atom));

  for(int i = 0; i < NUMBER_OF_ATOMS; i++){
    atoms[i] = (Atom*)malloc(sizeof(Atom));
  }

  hipMalloc((void**)&d_atoms, NUMBER_OF_ATOMS * sizeof(Atom*));
  hipMalloc(&d_atom_ref, NUMBER_OF_ATOMS * sizeof(Atom));
  //std::cout << "Blocks  = { " << blocks.x << ", " << blocks.y << ", " << blocks.z << "}\n";
  //std::cout << "thread_num  = { " << thread_num.x << ", " << thread_num.y << ", " << thread_num.z << "}\n"; 

  int number_of_blocks = (int)ceil(NUMBER_OF_ATOMS / (float)THREADS_PER_BLOCK);

  //std::cout << "Number of blocks = " << number_of_blocks << "\n";

  initalize_atoms<<< number_of_blocks, THREADS_PER_BLOCK>>>(d_atoms);

  hipDeviceSynchronize();

  start = std::chrono::system_clock::now();

  glutInit(&argc, argv);
  glutInitDisplayMode(GLUT_SINGLE | GLUT_RGB);

  // Position window at (80,80)-(480,380) and give it a title.
  glutInitWindowPosition(80, 80);
  glutInitWindowSize(STARTING_WIDTH, STARTING_HEIGHT);
  glutCreateWindow("A Simple Triangle");

  // Tell GLUT that whenever the main window needs to be repainted that it
  // should call the function display().
  glutDisplayFunc(display);

  glutReshapeFunc(reshape);

  glutTimerFunc(100, timer, 1);

  glutSpecialFunc(special);

  // Tell GLUT to start reading and processing events.  This function
  // never returns; the program only exits when the user closes the main
  // window or kills the process.
  glutMainLoop();


  hipFree(d_atoms);
  free(atoms);
}