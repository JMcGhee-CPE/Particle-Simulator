#include "hip/hip_runtime.h"
#pragma once

#include "glm/glm.hpp"
#include "glm/gtx/projection.hpp"

#include <windows.h>

#include <stdio.h>
#include <cstring>

#define ATOM_RADIUS 0.05
#define ATOM_RESOLUTION 12
#define NUMBER_OF_ATOMS 2048
#define ATOM_SPACING 2
#define STARTING_WIDTH 800
#define STARTING_HEIGHT 600
#define COLLISION_MOVE 0.1

#ifdef __HIPCC__
#define CUDA_CALLABLE_MEMBER __host__ __device__
#else
#define CUDA_CALLABLE_MEMBER
#endif

void DrawCircle(float cx, float cy, float r, int num_segments, const GLfloat* color)
{

    glPushMatrix();

      glColor3fv(color);

      glBegin(GL_POLYGON);
      for(int i = 0; i < num_segments; i++)
      {
          float theta = 2.0f * 3.1415926f * float(i) / float(num_segments);//get the current angle

          float x = r * cosf(theta);//calculate the x component
          float y = r * sinf(theta);//calculate the y component

          glVertex2f(x + cx, y + cy);//output vertex

      }
      glEnd();

    glPopMatrix();
}


class Atom {


public:

  glm::vec2 coords;
  glm::vec2 vel;
  glm::vec2 new_pos;
  glm::vec2 new_vel;
  bool update_vel = false;

  GLfloat color[3];


  CUDA_CALLABLE_MEMBER
  void setPosition(glm::vec2 position){
    coords = position;
  }

  CUDA_CALLABLE_MEMBER
  void setColor(GLfloat* new_color){
    color[0] = new_color[0];
    color[1] = new_color[1];
    color[2] = new_color[2];
  }

  CUDA_CALLABLE_MEMBER
  void setVelocity(glm::vec2 velocity){
    vel = velocity;
  }

  CUDA_CALLABLE_MEMBER
  glm::vec2 getVelocity(){
    return vel;
  }

  CUDA_CALLABLE_MEMBER
  glm::vec2 getPosition(){
    return coords;
  }

  CUDA_CALLABLE_MEMBER
  void move(float time){
    if (coords[1] >= (STARTING_HEIGHT / 100) || coords[1] <= -(STARTING_HEIGHT / 100)){
      vel *= glm::vec2(1,-1);
      if(coords[1] >= (STARTING_HEIGHT / 100)) {
        coords[1] = (STARTING_HEIGHT / 100);
      } else {
        coords[1] = -(STARTING_HEIGHT / 100);
      }
    }
    if (coords[0] >= (STARTING_WIDTH / 100) || coords[0] <= -(STARTING_WIDTH / 100)){
      vel *= glm::vec2(-1,1);
      if(coords[0] >= (STARTING_WIDTH / 100)) {
        coords[0] = (STARTING_WIDTH / 100);
      } else {
        coords[0] = -(STARTING_WIDTH / 100);
      }
    }
    glm::vec2 displacement = vel;
    displacement *= time;

    coords += displacement;
  }

  CUDA_CALLABLE_MEMBER
  void handleCollisions( Atom** potential_Collisions ){
    glm::vec2 new_vector(0,0);
    new_pos = coords;
    int num_collisions = 0;
    for(int i = 0; i < NUMBER_OF_ATOMS; i++){
      if((potential_Collisions[i] != this) && (abs(glm::distance( potential_Collisions[i]->coords, coords )) <= (2 * ATOM_RADIUS))){

        // Formula for elastic collisions https://williamecraver.wixsite.com/elastic-equations

        glm::vec2 position_vector = coords;
        position_vector -= potential_Collisions[i]->coords;

        glm::vec2 unit_vector = position_vector;
        unit_vector /= glm::length(position_vector);


        glm::vec2 offset_vector = unit_vector;
        offset_vector *= (2 * ATOM_RADIUS * COLLISION_MOVE);

        glm::vec2 velocity_difference = vel;
        velocity_difference -= potential_Collisions[i]->vel;

        float dotproduct = glm::dot(velocity_difference, position_vector);

        glm::vec2 contribution = position_vector;
        contribution *= -dotproduct / powf( glm::length(position_vector), 2);

        contribution += vel;

        new_vector += contribution;

        new_pos += offset_vector;

        num_collisions++;
      }
    }

    if(num_collisions > 0){
      //std::cout << "Num Collisions = " << num_collisions << " \n";
      new_vector /= num_collisions;
      //std::cout << "New Vector = {" << new_vector[0] << ", " << new_vector[1] << "} \n";
      new_vel = new_vector;
      //std::cout << "New vector - { " << new_vel[0] << ", " << new_vel[1] << " }\n";
      update_vel = true;
    }

    // Else no collisions, velocity unchanged

  }

  CUDA_CALLABLE_MEMBER
  void updateVelocity(){
    if(update_vel){
      //std::cout << "velocity updated!\n";
      vel = new_vel;
      coords = new_pos;
      update_vel = false;
    }
  }

  void draw(){

    // Set every pixel in the frame buffer to the current clear color.

    DrawCircle(coords[0], coords[1], ATOM_RADIUS, ATOM_RESOLUTION, color);
  }

  void print(){
    std::cout << "Coords: { " << coords[0] << ", " << coords[1] << " }; Vel: { " << vel[0] << ", " << vel[1] << "}\n";
  }

};